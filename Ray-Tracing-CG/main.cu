#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>

#include "PNGWriter.h"

using namespace std;


const int IMAGE_WIDTH = 256;
const int IMAGE_HEIGHT = 256;

const int THREAD_BLOCK_SIZE = 8;


__global__ void render(float* buffer, int width, int height) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= width || j >= height)
        return;

    int n = 3 * (i + j * width);
    buffer[n] = float(i) / (width - 1);
    buffer[n + 1] = float(height - j) / (height - 1);
    buffer[n + 2] = 0.25f;
}

int main()
{
    int buf_size = 3 * IMAGE_WIDTH * IMAGE_HEIGHT;
    float* colour_buf;
    hipMallocManaged((void**) &colour_buf, buf_size*sizeof(float));

    dim3 blocks(1 + IMAGE_WIDTH / THREAD_BLOCK_SIZE, 1 + IMAGE_HEIGHT / THREAD_BLOCK_SIZE);
    dim3 threads(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);

    render<<<blocks, threads>>>(colour_buf, IMAGE_WIDTH, IMAGE_HEIGHT);
    hipDeviceSynchronize();

    PNGWriter::write_file("image.png", IMAGE_WIDTH, IMAGE_HEIGHT, 3, colour_buf, buf_size);

    hipFree(colour_buf);
    

    return 0;
}