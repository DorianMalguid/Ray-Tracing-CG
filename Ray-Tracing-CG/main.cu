#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>

#include "PNGWriter.h"
#include "Vec3.h"

using namespace std;


const int IMAGE_WIDTH = 256;
const int IMAGE_HEIGHT = 256;

const int THREAD_BLOCK_SIZE = 8;


__global__ void render(Vec3* buffer, int width, int height) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;
    if (i >= width || j >= height)
        return;

    int n = i + j * width;
    buffer[n] = Vec3(float(i) / (width - 1),
                     float(height - j) / (height - 1),
                     0.25f);
}

int main()
{
    int buf_size = IMAGE_WIDTH * IMAGE_HEIGHT;
    Vec3* colour_buf;
    hipMallocManaged((void**) &colour_buf, buf_size*sizeof(Vec3));

    dim3 blocks(1 + IMAGE_WIDTH / THREAD_BLOCK_SIZE, 1 + IMAGE_HEIGHT / THREAD_BLOCK_SIZE);
    dim3 threads(THREAD_BLOCK_SIZE, THREAD_BLOCK_SIZE);

    render<<<blocks, threads>>>(colour_buf, IMAGE_WIDTH, IMAGE_HEIGHT);
    hipDeviceSynchronize();

    PNGWriter::write_file("image.png", IMAGE_WIDTH, IMAGE_HEIGHT, 3, colour_buf, buf_size);

    hipFree(colour_buf);
    

    return 0;
}