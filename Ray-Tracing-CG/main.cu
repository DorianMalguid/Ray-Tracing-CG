﻿#include "hip/hip_runtime.h"


#include <iostream>
#include <fstream>

using namespace std;


const int IMAGE_WIDTH = 256;
const int IMAGE_HEIGHT = 256;

void write_image_file(string file_name, float* colour_buf, int buf_size);

int main()
{
    int buf_size = 3 * IMAGE_WIDTH * IMAGE_HEIGHT;
    float* colour_buf = new float[buf_size];

    int k = 0;
    for (int j = IMAGE_HEIGHT - 1; j >= 0; j--) {
        for (int i = 0; i < IMAGE_WIDTH; i++)
        {
            colour_buf[k++] = float(i) / (IMAGE_WIDTH - 1);
            colour_buf[k++] = float(j) / (IMAGE_HEIGHT - 1);
            colour_buf[k++] = 0.25f;
        }
    }
    
    write_image_file("image.ppm", colour_buf, buf_size);

    return 0;
}



// Write the colour buffer into a ppm file
void write_image_file(string file_name, float* colour_buf, int buf_size) {
    ofstream out(file_name);

    out << "P3" << endl << IMAGE_WIDTH << " " << IMAGE_HEIGHT << endl << "255" << endl;

    int i = 0;

    while (i < buf_size) {
        int ir = int(255.999f * colour_buf[i++]);
        int ig = int(255.999f * colour_buf[i++]);
        int ib = int(255.999f * colour_buf[i++]);

        out << ir << " " << ig << " " << ib << endl;
    }
    out.close();
}